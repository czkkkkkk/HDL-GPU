#include "hip/hip_runtime.h"
#include "prefix_sum.h"

__global__ void BlockPrefix(int *a, int k, int n) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	for(int j = i * k + 1; j < i * k + k && j < n; ++j) {
		a[j] += a[j - 1];
	}
}

__global__ void Compute(int *a, int k, int n) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int id = i / k * 2 * k + k + i % k;
	if(id < n) {
		a[id] += a[id - id % k - 1];
	}
}

void PrefixSum(int *a, int n) {
	int block_size = 256;
	int threadsPerBlock = 256;
	BlockPrefix<<<(n + threadsPerBlock * block_size - 1) / (threadsPerBlock * block_size), threadsPerBlock>>>(a, block_size, n);
	for(int i = block_size; i < n; i *= 2) {
		int t = n / 2;
		int threadsPerBlock = 256;
		int numBlocks = (t + 256 - 1) / 256;
		Compute<<<numBlocks, threadsPerBlock>>>(a, i, n);
	}
}
