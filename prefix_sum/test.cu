#include "prefix_sum.h"

#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>

int main() {
	int n = 1 << 20;
	int *a, *da;
	a = (int*)malloc(n * sizeof(int));
	for(int i = 0; i < n; ++i) a[i] = 1;
	if(hipSuccess != hipMalloc((void **)&da, n * sizeof(int))) {
		puts("Error");
		exit(1);
	}
	hipMemcpy(da, a, n * sizeof(int), hipMemcpyHostToDevice);
	PrefixSum(da, n);
	hipMemcpy(a, da, n * sizeof(int), hipMemcpyDeviceToHost);
	for(int i = 0; i < n; ++i) {
		if(a[i] != i + 1) {
			puts("Error");
			exit(1);
		}
	}
	hipFree(da);

	free(a);
	return 0;
}
