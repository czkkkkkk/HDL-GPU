#include "vec_add.h"

#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>

int main() {
	int n = 256;
	int *a, *b, *c, *da, *db, *dc;
	a = (int*)malloc(n * sizeof(int));
	b = (int*)malloc(n * sizeof(int));
	c = (int*)malloc(n * sizeof(int));
	for(int i = 0; i < n; ++i) a[i] = b[i] = 1;
	if(hipSuccess != hipMalloc((void **)&da, n * sizeof(int))) {
		puts("Error");
	}
	hipMalloc((void **)&db, n * sizeof(int));
	hipMalloc((void **)&dc, n * sizeof(int));
	hipMemcpy(da, a, n * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(db, b, n * sizeof(int), hipMemcpyHostToDevice);
	VecAdd<<<1, n>>>(da, db, dc, n);
	hipMemcpy(c, dc, n * sizeof(int), hipMemcpyDeviceToHost);
	for(int i = 0; i < n; ++i) {
		if(c[i] != 2) {
			printf("Error\n");
			exit(1);
		}
	}
	hipFree(da);
	hipFree(db);
	hipFree(dc);

	free(a);
	free(b);
	free(c);
	return 0;
}
